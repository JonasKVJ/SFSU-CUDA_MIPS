#include "hip/hip_runtime.h"
/****                                                                           
     File: findRedsDriver.cu
     Date: 5/3/2018
     By: Bill Hsu
****/
/*
 * How to compile and execute:
 * source ~whsu/lees.bash_profile
 * nvcc findRedsGPU.cu -o frgpu -lm -Wno-deprecated-gpu-targets
 * ./frgpu
 * Submission by: Jonas Vinter-Jensen, 912941515
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUMPARTICLES 1024
#define NEIGHBORHOOD .05
#define THREADSPERBLOCK 4

void initPos(float*);

float findDistance(float*, int, int);

__device__ float findDistanceGPU(float*, int, int);

void dumpResults(int index[]);

__global__ void findRedsGPU(float* p, int* numI);

int main(int argc, const char* argv[])
{
    hipEvent_t start, stop;
    float time;

    float* pos;
    float* dpos;
    int* numReds;
    int* dnumReds;

    pos = (float*) malloc(NUMPARTICLES * 4 * sizeof(float));
    numReds = (int*) malloc(NUMPARTICLES * sizeof(int));

    initPos(pos);

    // your code to allocate device arrays for pos and numReds go here
    hipMalloc((void**) &dpos, NUMPARTICLES * 4 * sizeof(float));
    hipMalloc((void**) &dnumReds, NUMPARTICLES * sizeof(int));
    hipMemcpy(dpos, pos, NUMPARTICLES * 4 * sizeof(float), hipMemcpyHostToDevice); //dpos = copy(pos)

    // create timer events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0); //(event, stream)

    /* invoke kernel findRedsGPU here */
    findRedsGPU<<<NUMPARTICLES/THREADSPERBLOCK, THREADSPERBLOCK>>>(dpos, dnumReds);

    hipDeviceSynchronize();

    // your code to copy results to numReds[] go here
    hipMemcpy(numReds, dnumReds, NUMPARTICLES * sizeof(int), hipMemcpyDeviceToHost); //numReds = copy(dnumReds)

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); //waits for record event to complete
    hipEventElapsedTime(&time, start, stop);

    printf("Elapsed time = %f\n", time);

    dumpResults(numReds);

    free(pos);
    hipFree(dpos);
    free(numReds);
    hipFree(dnumReds);

    return 0;
}

void initPos(float* p)
{
    // your code for initializing pos goes here
    int i;
    for (i = 0; i < NUMPARTICLES; i++)
    {
        p[i * 4] = rand() / (float) RAND_MAX; //p.x
        p[i * 4 + 1] = rand() / (float) RAND_MAX; //p.y
        p[i * 4 + 2] = rand() / (float) RAND_MAX; //p.z

        int colorChoice;
        colorChoice = random() % 3;
        if (colorChoice == 0)
        {
            p[i * 4 + 3] = 0xff0000; //p.color = red
        }
        else if (colorChoice == 1)
        {
            p[i * 4 + 3] = 0x00ff00; //p.color = green
        }
        else
        {
            p[i * 4 + 3] = 0x0000ff; //p.color = blue
        }
    }
}

__device__ float findDistanceGPU(float* p, int i, int j)
{
    // your code for calculating distance for particle i and j
    float dx, dy, dz;

    dx = p[i * 4] - p[j * 4]; //x2-x1
    dy = p[i * 4 + 1] - p[j * 4 + 1]; //y2-y1
    dz = p[i * 4 + 2] - p[j * 4 + 2]; //z2-z1

    return (sqrt(dx * dx + dy * dy + dz * dz));
}

__global__ void findRedsGPU(float* p, int* numI)
{
    // your code for counting red particles goes here
    int k;
    float distance;

    int p2_num = blockDim.x*blockIdx.x + threadIdx.x;
    for (k = 0; k < NUMPARTICLES; k++)
    {
        /*Every time a new (blockId, threadId) permutation occurs, initialize number of red particles near particle
         * k to 0 for every first k-loop iteration of the pairs.*/
        if(k==0)
        {
            numI[p2_num] = 0;
        }
        if (k != p2_num)
        {
            /* calculate distance between particles k, p2_num */
            distance = findDistanceGPU(p, k, p2_num);

            /* if distance < r and color is red, increment count */
            if (distance < NEIGHBORHOOD && p[p2_num * 4 + 3] == 0xff0000)
            {
                numI[k]++;
            }
        }
    }

}

void dumpResults(int index[])
{
    int i;
    FILE* fp;

    fp = fopen("./dump.out", "w");

    for (i = 0; i < NUMPARTICLES; i++)
    {
        fprintf(fp, "%d %d\n", i, index[i]);
    }
    fclose(fp);
}
